
#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>

using namespace std;

const int MAX_COEFF = 103;

void random_polynomial(int* p,  int n)
{
    for (int i=0; i<n; i++) {
        p[i] = rand() % MAX_COEFF;
    }
}

__global__ void calculate_products(int *prods, int *x, int *y, size_t n) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    prods[index] = (x[blockIdx.x] * y[threadIdx.x]) % MAX_COEFF;
}


__global__ void reduce_polynomial(int *prods, int *ans, size_t n)
{
    // combine like terms
    int i, j;
    if (blockIdx.x <= (2*n-2)/2)
    {
        i = blockIdx.x, j = 0;
    }
    else
    {
        i = n-1, j = (blockIdx.x % n) + 1;
    }
    while (i >= 0 && j < n)
    {
        ans[blockIdx.x] = (ans[blockIdx.x] + prods[i*n + j]) % MAX_COEFF;
        i--;
        j++;
    }
}

int main() {
    srand(time(NULL));
    const int n = 1024; // 2^10
    int *X = NULL;
    int *Y = NULL;
    int *P = NULL; // products
    int *Poly = NULL;
    X = new int[n];
    Y = new int[n];
    P = new int[n*n];
    Poly = new int[2*n-1];

    // Initialize values
    random_polynomial(X, n);
    random_polynomial(Y, n);

    for (int i = 0; i < n*n; i++)
    {
        P[i] = 0;
    }

    for (int i = 0; i < 2*n-1; i++)
    {
        Poly[i] = 0;
    }

    // Products
	int *Xd, *Yd, *Pd;
    hipMalloc((void **)&Xd, sizeof(int)*n);
    hipMalloc((void **)&Yd, sizeof(int)*n);
    hipMalloc((void **)&Pd, sizeof(int)*n*n);

	hipMemcpy(Xd, X, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Yd, Y, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Pd, P, sizeof(int)*n*n, hipMemcpyHostToDevice);

    calculate_products<<<n, n>>>(Pd, Xd, Yd, n);

    int *Polyd;
    hipMalloc((void **)&Polyd, sizeof(int)*2*n-1);

    hipMemcpy(Polyd, Poly, sizeof(int)*2*n-1, hipMemcpyHostToDevice);

    // Reduction kernel
    reduce_polynomial<<<2*n-1, 1>>>(Pd, Polyd, n);
    hipMemcpy(Poly, Polyd, sizeof(int)*2*n-1, hipMemcpyDeviceToHost);

    // Print input, output
    for (int i = 0; i < n; ++i) printf("%2d ", X[i]);
    printf("\n\n");
    for (int i = 0; i < n; ++i) printf("%2d ", Y[i]);
    printf("\n\n");
    for (int i = 0; i < 2*n-1; ++i) printf("%2d ", Poly[i]);
    printf("\n\n");
    
    delete [] X;
    delete [] Y;
    delete [] P;
    delete [] Poly;

	
	hipFree(Xd);
    hipFree(Yd);
    hipFree(Pd);
    hipFree(Polyd);
	
	return 0;
}