// CS 4402 - Dana Zagar - 250790176

#include <hip/hip_runtime.h>
#include <cstdio>
#include <ctime>

using namespace std;

// A small prime number to prevent overflow and make verification feasible.
const int MAX_COEFF = 103;

// Print polynomial output.
void print_polynomial(int* poly, int range)
{
    for (int i = 0; i < range; i++) 
    {
        printf("%2d ", poly[i]);
    }
    printf("\n\n");
}

// Generates a random polynomial of size n.
void random_polynomial(int* p,  int n)
{
    for (int i=0; i<n; i++) {
        p[i] = rand() % MAX_COEFF;
    }
}

// Serial C function to find reduced polynomial product.
// For verification purposes.
void multiply_polynomials_serial(int *x, int *y, int size, int *ans)
{
    for (int i = 0; i < size; i++)
    {
        for (int j = 0; j < size; j++)
        {
            ans[i+j] = (ans[i+j] + x[i] * y[j]) % MAX_COEFF;
        }
    }
}

// First CUDA kernel to calculate the product terms over two given polynomials
// of size n, given n thread-blocks and n threads per.
__global__ void calculate_products(int *prods, int *x, int *y, size_t n) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    prods[index] = (x[blockIdx.x] * y[threadIdx.x]) % MAX_COEFF;
}

// Second CUDA kernel to reduce the products by combining like terms on each
// diagonal of the "2d" product matrix.
__global__ void reduce_polynomial(int *prods, int *ans, size_t n)
{
    int i, j;
    
    // Envision the product array as a 2d matrix tilted like a diamond.
    // Each block represents a row of the diamond, i.e. a diagonal.
    // If the block index is within the first half of the diamond, the
    // block index dictates the row index.
    if (blockIdx.x <= (2*n-2)/2)
    {
        i = blockIdx.x, j = 0;
    }
    // Otherwise, the block index dictates the column index.
    else
    {
        i = n-1, j = (blockIdx.x % n) + 1;
    }

    // Sum over the diagonal given by the block index.
    while (i >= 0 && j < n)
    {
        ans[blockIdx.x] = (ans[blockIdx.x] + prods[i*n + j]) % MAX_COEFF;
        i--;
        j++;
    }
}

int main() {
    srand(time(NULL));
    int exponent;

    // Input the number of terms.
    printf("Input the desired number of terms in the polynomials. Enter an exponent on 2 [valid from 1-10] to define 2^input terms: ");
    scanf("%d", &exponent);

    if (exponent < 1 || exponent > 10)
    {
        printf("Invalid input. Program will terminate.\n\n");
        return 0;
    }

    int n = 1 << exponent; // Number of terms is 2^exponent.
    printf("%d terms; input polynomials are of degree %d.\n\n", n, n-1);

    int *X = NULL; // First polynomial of degree n-1.
    int *Y = NULL; // Second polynomial of degree n-1.
    int *P = NULL; // Interim products.
    int *Poly = NULL; // Final.
    int *PolyV = NULL; // Verification answer.
    X = new int[n];
    Y = new int[n];
    P = new int[n*n];
    Poly = new int[2*n-1];
    PolyV = new int[2*n-1];

    // Initialize values.
    random_polynomial(X, n);
    random_polynomial(Y, n);

    for (int i = 0; i < n*n; i++)
    {
        P[i] = 0;
    }

    for (int i = 0; i < 2*n-1; i++)
    {
        Poly[i] = 0;
        PolyV[i] = 0;
    }

    // Step 1: Calculating products.
	int *Xd, *Yd, *Pd;
    hipMalloc((void **)&Xd, sizeof(int)*n);
    hipMalloc((void **)&Yd, sizeof(int)*n);
    hipMalloc((void **)&Pd, sizeof(int)*n*n);

	hipMemcpy(Xd, X, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Yd, Y, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Pd, P, sizeof(int)*n*n, hipMemcpyHostToDevice);

    calculate_products<<<n, n>>>(Pd, Xd, Yd, n);

    // Step 2: Reducing like terms.
    int *Polyd;
    hipMalloc((void **)&Polyd, sizeof(int)*2*n-1);

    hipMemcpy(Polyd, Poly, sizeof(int)*2*n-1, hipMemcpyHostToDevice);

    reduce_polynomial<<<2*n-1, 1>>>(Pd, Polyd, n);
    hipMemcpy(Poly, Polyd, sizeof(int)*2*n-1, hipMemcpyDeviceToHost);

    // Print input, output.
    printf("CUDA Program Output\n\n");
    print_polynomial(X, n);
    print_polynomial(Y, n);
    print_polynomial(Poly, 2*n-1);

    // Step 3: Verify using serial C function.
    printf("Verification with Serial C Output\n\n");
    multiply_polynomials_serial(X, Y, n, PolyV);
    print_polynomial(PolyV, 2*n-1);
    
    // Free memory.
    delete [] X;
    delete [] Y;
    delete [] P;
    delete [] Poly;
    delete [] PolyV;
	
	hipFree(Xd);
    hipFree(Yd);
    hipFree(Pd);
    hipFree(Polyd);
	
	return 0;
}
