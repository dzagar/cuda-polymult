
#include <hip/hip_runtime.h>
#include <cstdio>

using namespace std;

const int MAX_COEFF = 103;

void random_polynomial(int* p,  int n)
{
    for (int i=0; i<n; i++) {
        p[i] = rand() % MAX_COEFF;
    }
}

__global__ void calculate_products(int *prods, int *x, int *y, size_t n) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    prods[index] = x[blockIdx.x] * y[threadIdx.x];
}


__global__ void reduce_polynomial(int *prods, int *ans, size_t n)
{
    // combine like terms
    int i, j;
    if (blockIdx.x <= (2*n-2)/2)
    {
        i = blockIdx.x, j = 0;
    }
    else
    {
        i = n-1, j = (blockIdx.x % n) + 1;
    }
    while (i >= 0 && j < n)
    {
        ans[blockIdx.x] += prods[i*n + j];
        i--;
        j++;
    }
}

int main() {
    const int n = 2048;    
    int *X = NULL;
    int *Y = NULL;
    int *P = NULL; // products
    int *Poly = NULL;
    X = new int[n];
    Y = new int[n];
    P = new int[n*n];
    Poly = new int[2*n-1];

    random_polynomial(X, n);
    random_polynomial(Y, n);

    for (int i = 0; i < n*n; i++)
    {
        P[i] = 0;
    }

    // Products
	
	int *Xd, *Yd, *Pd;
	hipMalloc((void **)&Xd, sizeof(int)*n);
    hipMalloc((void **)&Yd, sizeof(int)*n);
    hipMalloc((void **)&Pd, sizeof(int)*n*n);

	hipMemcpy(Xd, X, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Yd, Y, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(Pd, P, sizeof(int)*n*n, hipMemcpyHostToDevice);

	calculate_products<<<n, n>>>(Pd, Xd, Yd, n);
    hipMemcpy(P, Pd, sizeof(int)*n*n, hipMemcpyDeviceToHost);
    for (int i = 0; i < 2*n-1; ++i) printf("%2d ", P[i]);
    // Sums to final polynomial

    int *Polyd;
    hipMalloc((void **)&Polyd, sizeof(int)*2*n-1);

    hipMemcpy(Pd, P, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(Polyd, Poly, sizeof(int)*2*n-1, hipMemcpyHostToDevice);

    // START REDUCTION KERNEL HERE AND JUST FOR-LOOP THRU THE BLOCK
    reduce_polynomial<<<2*n-1, 1>>>(Pd, Polyd, n);
    hipMemcpy(Poly, Polyd, sizeof(int)*2*n-1, hipMemcpyDeviceToHost);

    // Print input, output
    // for (int i = 0; i < n; ++i) printf("%2d ", X[i]);
    // printf("\n\n");
    // for (int i = 0; i < n; ++i) printf("%2d ", Y[i]);
    // printf("\n\n");
    // for (int i = 0; i < 2*n-1; ++i) printf("%2d ", Poly[i]);
	// printf("\n\n");
	
	hipFree(Xd);
    hipFree(Yd);
    hipFree(Pd);
    hipFree(Polyd);
	
	return 0;
}
